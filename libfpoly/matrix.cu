#include "hip/hip_runtime.h"

/*
 * compiles on elephanttest using
 * nvcc --compiler-options '-fPIC' -o libfpoly.so --shared matrix.cu
 */
 
#include <hip/hip_runtime.h>

#define aref(mat, row, col, n) (mat[(col)*(n) + (row)])

/* do the echelon operation */
__device__ int ffge(int *mat, int *vec, int n);

/* launch the threads on the GPU */
__global__ int ffge_gpu(int *mats, int *vecs, int num, int n);

/* pivoting rows */
__device__ int pivot(int *a, int *b, int i, int n);

/* the entry point for calling */
void ffge_list (int *mats, int *vecs, int num, int n);


/* a = matrix, b = vector */
__device__ int ffge(int *a, int *b, int n) {
    int i, j, k;

	for(i = 0; i < n - 1; ++i) {
		if (aref(a, i, i, n) == 0) {
			/* need to pivot */
			if (pivot(a, b, i, n)) {
				/* can't pivot! probably will divide by zero so return an error code */
				return 1;
			}
		}
		
		for(j = i + 1; j < n; ++j) {
			b[j] = aref(a, i, i, n)*b[j] - aref(a, j, i, n)*b[i];
			if (i > 0) {
				b[j] = b[j] / aref(a, i-1, i-1, n);
			}

			for(k = i + 1; k < n; ++k){
				aref(a, j, k, n) = aref(a, i, i, n)*aref(a, j, k, n) - aref(a, j, i, n)*aref(a, i, k, n);
				if (i > 0) {
					aref(a, j, k, n) = aref(a, j, k, n) / aref(a, i-1, i-1, n);
				}
			}

			aref(a, j, i, n) = 0;
		}
	}

	return 0;
}

__device__ int pivot(int *a, int *b, int i, int n) {
	int row, col, tmp;

	int err = 1;
	for(row=i; row < n; ++row) {
		if (aref(a, row, i, n) != 0) {
			/* found a row with non-zero pivot element */
			for(col=0; col < n; ++col) {
				tmp = aref(a, i, col, n);
				aref(a, i, col, n) = aref(a, row, col, n); 
				aref(a, row, col, n) = tmp;

				tmp = b[i];
				b[i] = b[row];
				b[row] = tmp;
			}
			
			err = 0;
			break;
		}
	}
	
	return err;
}

__global__ void ffge_gpu(int *mats, int *vecs, int num, int n) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int mat_size = sizeof(int)*n*n;
	int vec_size = sizeof(int)*n;
	
	if (i < num) {
		ffge(mats + i*mat_size, vecs + i*vec_size, n);
	}
}

int ffge_list(int *h_mats, int *h_vecs, int num, int n) {
	int mat_size = sizeof(int)*n*n;
	int vec_size = sizeof(int)*n;
	 int *d_mats, *d_vecs;
	
	 /* hipMalloc the arrays and copy over to the GPU */
	 hipError_t err = hipSuccess;
	 
	 err = hipMalloc((void **)&d_mats, num*mat_size);
	 if (err != hipSuccess) {
	 	return 1;
	 }
	 
	 err = hipMalloc((void **)&d_vecs, num*vec_size);
	 if (err != hipSuccess) {
	 	return 1;
	 }

	 /* now copy the data over */
	 err = hipMemcpy(d_mats, h_mats, num*mat_size, hipMemcpyHostToDevice);
	 if (err != hipSuccess) {
	 	return 1;
	 }
	 
	 err = hipMemcpy(d_vecs, h_vecs, num*vec_size, hipMemcpyHostToDevice);
	 if (err != hipSuccess) {
	 	return 1;
	 }

	 /* do the computation on the GPU */
	 ffge_gpu<<<1,num>>>(d_mats, d_vecs, num, n);

	 /* now copy the results back to the host memory */

	 err = hipMemcpy(h_mats, d_mats, num*mat_size, hipMemcpyDeviceToHost);
	 if (err != hipSuccess) {
	 return 1;
	 }
	 
	 err = hipMemcpy(h_vecs, d_vecs, num*vec_size, hipMemcpyDeviceToHost);
	 if (err != hipSuccess) {
	 	return 1;
	}

	 err = hipFree(d_mats);
	 if (err != hipSuccess) {
	 return 1;
	 }
	 
	 err = hipFree(d_vecs);
	 if (err != hipSuccess){
	 	return 1;
	}

	 return 0;
}

	
