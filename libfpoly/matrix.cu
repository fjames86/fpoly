
/*
 * compiles on elephanttest using
 * nvcc --compiler-options '-fPIC' -o libfpoly.so --shared matrix.cu
 */
 
#include <hip/hip_runtime.h>

/* a = matrix, b = vector */
__device__ void ffge(int *a, int *b, int n) {
	int i, j, k;

	for(i = 0; i < n - 1; ++i) {
		for(j = i + 1; j < n; ++j) {
			b[j] = a[i*n + i]*b[j] - a[j*n + i]*b[i];
			if (i > 0) {
				b[j] = b[j] / a[(i-1)*n + i-1];
			}

			for(k = i + 1; k < n; ++k){
				a[j*n + k] = a[i*n + i]*a[j*n + k] - a[j*n + i]*a[i*n + k];
				if (i > 0) {
					a[j*n + k] = a[j*n + k] / a[(i-1)*n + i-1];
				}
			}

			a[j*n + i] = 0;
		}
	}
}

__global__ void ffge_gpu(int *mats, int *vecs, int num, int n) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int mat_size = sizeof(int)*n*n;
	int vec_size = sizeof(int)*n;
	
	if (i < num) {
		ffge(mats + i*mat_size, vecs + i*vec_size, n);
	}
}

int ffge_list(int *h_mats, int *h_vecs, int num, int n) {
	int mat_size = sizeof(int)*n*n;
	int vec_size = sizeof(int)*n;
	 int *d_mats, *d_vecs;
	
	 /* cudaMalloc the arrays and copy over to the GPU */
	 hipError_t err = hipSuccess;
	 
	 err = hipMalloc((void **)&d_mats, num*mat_size);
	 if (err != hipSuccess) {
	 	return 1;
	 }
	 
	 err = hipMalloc((void **)&d_vecs, num*vec_size);
	 if (err != hipSuccess) {
	 	return 1;
	 }

	 /* now copy the data over */
	 err = hipMemcpy(d_mats, h_mats, num*mat_size, hipMemcpyHostToDevice);
	 if (err != hipSuccess) {
	 	return 1;
	 }
	 
	 err = hipMemcpy(d_vecs, h_vecs, num*vec_size, hipMemcpyHostToDevice);
	 if (err != hipSuccess) {
	 	return 1;
	 }

	 /* do the computation on the GPU */
	 ffge_gpu<<<1,num>>>(d_mats, d_vecs, num, n);

	 /* now copy the results back to the host memory */

	 err = hipMemcpy(h_mats, d_mats, num*mat_size, hipMemcpyDeviceToHost);
	 if (err != hipSuccess) {
	 return 1;
	 }
	 
	 err = hipMemcpy(h_vecs, d_vecs, num*vec_size, hipMemcpyDeviceToHost);
	 if (err != hipSuccess) {
	 	return 1;
	}

	 err = hipFree(d_mats);
	 if (err != hipSuccess) {
	 return 1;
	 }
	 
	 err = hipFree(d_vecs);
	 if (err != hipSuccess){
	 	return 1;
	}

	 return 0;
}

	
